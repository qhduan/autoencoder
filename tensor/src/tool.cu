#include "../inc/tool.h"


#include <iostream>

#include <hipblas.h>

namespace tensor {

hipEvent_t start, stop;
float timeValue;

void StartTimer () {
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord( start, 0 );
}

float EndTimer () {
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  hipEventElapsedTime( &timeValue, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  return timeValue;
}



}
