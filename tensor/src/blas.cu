#include "../inc/blas.h"

#include <stdexcept>
#include <iostream>
#include <string>

#include <cblas.h>

namespace tensor {

template<>
BLAS<CPU>* BLAS<CPU>::instance_ = NULL;

template<>
BLAS<GPU>* BLAS<GPU>::instance_ = NULL;

inline CBLAS_TRANSPOSE GetCblasTrans(bool t) {
	return t ? CblasTrans : CblasNoTrans;
}

inline hipblasOperation_t GetCudablasTrans(bool t) {
	return t ? HIPBLAS_OP_T : HIPBLAS_OP_N;
}

// CPU

template<>
BLAS<CPU>::BLAS () {
}
template<>
BLAS<CPU>::~BLAS () {
}

// CPU float

template<> template<>
float BLAS<CPU>::asum<float> (int n, const float* x, int incx) {
	return cblas_sasum(n, x, incx);
}

template<> template<>
void BLAS<CPU>::copy<float> (int n, const float* x, int incx, float* y, int incy) {
	cblas_scopy(n, x, incx, y, incy);
}

template<> template<>
float BLAS<CPU>::dot<float> (int n, const float* x, int incx, const float* y, int incy) {
	return cblas_sdot(n, x, incx, y, incy);
}

template<> template<>
float BLAS<CPU>::nrm2<float> (int n, const float* x, int incx) {
	return cblas_snrm2(n, x, incx);
}

template<> template<>
void BLAS<CPU>::ger<float> (int m, int n, float alpha, const float* x, int incx,
  const float* y, int incy, float *A, const int lda) {
	cblas_sger(CblasColMajor, m, n, alpha, x, incx, y, incy, A, lda);
}

template<> template<>
void BLAS<CPU>::gemv<float> (bool transa, int m, int n, float alpha, const float* A, int lda,
  const float* x, int incx, float beta, float* y, int incy) {
  cblas_sgemv(CblasColMajor, GetCblasTrans(transa), m, n,
    alpha, A, lda, x, incx, beta, y ,incy);
}

template<> template<>
void BLAS<CPU>::gemm<float> (bool transa, bool transb, int m, int n, int k, float alpha,
	const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc ) {
	cblas_sgemm(CblasColMajor, GetCblasTrans(transa), GetCblasTrans(transb), m, n, k, alpha,
		A, lda, B, ldb, beta,C, ldc);
}

// CPU double

template<> template<>
double BLAS<CPU>::asum<double> (int n, const double* x, int incx) {
	return cblas_dasum(n, x, incx);
}

template<> template<>
void BLAS<CPU>::copy<double> (int n, const double* x, int incx, double* y, int incy) {
	cblas_dcopy(n, x, incx, y, incy);
}

template<> template<>
double BLAS<CPU>::dot<double> (int n, const double* x, int incx, const double* y, int incy) {
	return cblas_ddot(n, x, incx, y, incy);
}

template<> template<>
double BLAS<CPU>::nrm2<double> (int n, const double* x, int incx) {
	return cblas_dnrm2(n, x, incx);
}

template<> template<>
void BLAS<CPU>::ger<double> (int m, int n, double alpha, const double* x, int incx,
  const double* y, int incy, double *A, const int lda) {
	cblas_dger(CblasColMajor, m, n, alpha, x, incx, y, incy, A, lda);
}

template<> template<>
void BLAS<CPU>::gemv<double> (bool transa, int m, int n, double alpha, const double* A, int lda,
  const double* x, int incx, double beta, double* y, int incy) {
  cblas_dgemv(CblasColMajor, GetCblasTrans(transa), m, n,
    alpha, A, lda, x, incx, beta, y ,incy);
}

template<> template<>
void BLAS<CPU>::gemm<double> (bool transa, bool transb, int m, int n, int k, double alpha, \
	const double *A, int lda, const double *B, int ldb, double beta, double *C, int ldc ) {
	cblas_dgemm(CblasColMajor, GetCblasTrans(transa), GetCblasTrans(transb), m, n, k, alpha,\
		A, lda, B, ldb, beta, C, ldc);
}

// GPU

const char* cublasGetErrorString (hipblasStatus_t err) {
  switch (err) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
		case HIPBLAS_STATUS_NOT_SUPPORTED:
			return "HIPBLAS_STATUS_NOT_SUPPORTED";
		case HIPBLAS_STATUS_UNKNOWN:
			return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "UNKNOWN";
}

#define CUBLAS_CALL(str, x) hipblasStatus_t err = (x); if (err != HIPBLAS_STATUS_SUCCESS) { std::cerr<<"error: "<<cublasGetErrorString(err)<<std::endl; throw std::runtime_error(str); }

template<>
BLAS<GPU>::BLAS () {
	CUBLAS_CALL("hipblasCreate", hipblasCreate(&cublas_handle_));
}
template<>
BLAS<GPU>::~BLAS () {
	hipblasDestroy(cublas_handle_);
}

// GPU float

template<> template<>
float BLAS<GPU>::asum<float> (int n, const float* x, int incx) {
	float result;
	CUBLAS_CALL("hipblasSasum", hipblasSasum(cublas_handle_, n, x, incx, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
void BLAS<GPU>::copy<float> (int n, const float* x, int incx, float* y, int incy) {
	CUBLAS_CALL("hipblasScopy", hipblasScopy(cublas_handle_, n, x, incx, y, incy));
  hipDeviceSynchronize();
}

template<> template<>
float BLAS<GPU>::dot<float> (int n, const float* x, int incx, const float* y, int incy) {
	float result;
	CUBLAS_CALL("hipblasSdot", hipblasSdot(cublas_handle_, n, x, incx, y, incy, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
float BLAS<GPU>::nrm2<float> (int n, const float* x, int incx) {
	float result;
	CUBLAS_CALL("hipblasSnrm2", hipblasSnrm2(cublas_handle_, n, x, incx, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
void BLAS<GPU>::ger<float> (int m, int n, float alpha, const float* x, int incx,
  const float* y, int incy, float *A, const int lda) {
	CUBLAS_CALL("hipblasSger", hipblasSger(cublas_handle_, m, n, &alpha, x, incx, y, incy, A, lda));
  hipDeviceSynchronize();
}

template<> template<>
void BLAS<GPU>::gemv<float> (bool transa, int m, int n, float alpha, const float* A, int lda,
  const float* x, int incx, float beta, float* y, int incy) {
  CUBLAS_CALL("hipblasSgemv", hipblasSgemv(cublas_handle_, GetCudablasTrans(transa), m, n, &alpha, A, lda, x, incx, &beta, y ,incy));
  hipDeviceSynchronize();
}

template<> template<>
void BLAS<GPU>::gemm<float> (bool transa, bool transb, int m, int n, int k, float alpha,
	const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc ) {
	CUBLAS_CALL("hipblasSgemm", hipblasSgemm(cublas_handle_, GetCudablasTrans(transa), GetCudablasTrans(transb), m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
  hipDeviceSynchronize();
}

// GPU double

template<> template<>
double BLAS<GPU>::asum<double> (int n, const double* x, int incx) {
	double result;
	CUBLAS_CALL("hipblasDasum", hipblasDasum(cublas_handle_, n, x, incx, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
void BLAS<GPU>::copy<double> (int n, const double* x, int incx, double* y, int incy) {
	CUBLAS_CALL("hipblasDcopy", hipblasDcopy(cublas_handle_, n, x, incx, y, incy));
  hipDeviceSynchronize();
}

template<> template<>
double BLAS<GPU>::dot<double> (int n, const double* x, int incx, const double* y, int incy) {
	double result;
	CUBLAS_CALL("hipblasDdot", hipblasDdot(cublas_handle_, n, x, incx, y, incy, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
double BLAS<GPU>::nrm2<double> (int n, const double* x, int incx) {
	double result;
	CUBLAS_CALL("hipblasDnrm2", hipblasDnrm2(cublas_handle_, n, x, incx, &result));
  hipDeviceSynchronize();
	return result;
}

template<> template<>
void BLAS<GPU>::ger<double> (int m, int n, double alpha, const double* x, int incx,
  const double* y, int incy, double *A, const int lda) {
	CUBLAS_CALL("hipblasDger", hipblasDger(cublas_handle_, m, n, &alpha, x, incx, y, incy, A, lda));
  hipDeviceSynchronize();
}

template<> template<>
void BLAS<GPU>::gemv<double> (bool transa, int m, int n, double alpha, const double* A, int lda,
  const double* x, int incx, double beta, double* y, int incy) {
  CUBLAS_CALL("hipblasDgemv", hipblasDgemv(cublas_handle_, GetCudablasTrans(transa), m, n, &alpha, A, lda, x, incx, &beta, y ,incy));
  hipDeviceSynchronize();
}

template<> template<>
void BLAS<GPU>::gemm<double> (bool transa, bool transb, int m, int n, int k, double alpha,
	const double *A, int lda, const double *B, int ldb, double beta, double *C, int ldc ) {
	CUBLAS_CALL("hipblasDgemm", hipblasDgemm(cublas_handle_, GetCudablasTrans(transa), GetCudablasTrans(transb), m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
  hipDeviceSynchronize();
}


}
