#include "../inc/mem.h"

#include <cstdlib>
#include <stdexcept>
#include <iostream>

namespace tensor {

#define CUDA_MEM_CALL(x) hipError_t err = (x); if (err != hipSuccess) { std::cerr<<"error: "<<hipGetErrorString(err)<<std::endl; throw std::bad_alloc(); }

template<>
void* Mem<CPU>::malloc(int bytes) {
	void* result;
	CUDA_MEM_CALL(hipHostMalloc((void**)&result, bytes));
	return result;
}

template<>
void Mem<CPU>::free(void* ptr) {
  CUDA_MEM_CALL(hipHostFree(ptr));
}


template<>
void* Mem<GPU>::malloc(int bytes) {
	void* result;
	CUDA_MEM_CALL(hipMalloc((void**)&result, bytes));
	return result;
}


template<>
void Mem<GPU>::free(void* ptr) {
  CUDA_MEM_CALL(hipFree(ptr));
}

}
